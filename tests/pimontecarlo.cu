#include "hip/hip_runtime.h"

import cobre.system;
import cobre.float;
import cobre.string;

float, int rand(int st) {
	st = st * 4005 + 165;
	while (st >= 65536) {
		st = st - 65536;
	}
	return itof(st / 65536.0), st;
}

void main () {
	int st = 1234;
	int count = 10000;
	int inside = 0;
	int i = 0;

	float start = clock();
	while (i < count) {
		float x, y;
		x, st = rand(st);
		y, st = rand(st);
		if (x*x + y*y <= 1.0) {
			inside = inside+1;
		}
		i = i+1;
	}
	float fin = clock();
	float time = fin - start;

	float pi = itof((inside/count)*4);
	print("PI: " + ftos(pi) + " in " + ftos(time) + "s");
}
