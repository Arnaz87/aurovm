#include "hip/hip_runtime.h"

/**< Add hoc utility module for working with the system */

/** Prints a line to standard output */
void print (string);

/** Reads a line from standard input */
string read ();

/** Executes a command in the OS shell and returns its output */
string cmd (string);

/** Executes a command in the OS shell and returns its error code */
int exec (string);

/** Returns the time spent by the process in seconds */
float clock ();

/** Reads the contents of a file into a byte array */
byte[] readfile (string filename);

/** Writes a byte array into a file */
void writefile (string filename, byte[] content);
