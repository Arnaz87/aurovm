#include "hip/hip_runtime.h"

/** Prints a line to standard output */
void print (string);

/** Reads a line from standard input */
string read ();

/** Executes a command in the OS shell, returns the error code and the standard output */
int, string cmd (string);

/** Returns the time spent by the process in seconds */
float clock ();