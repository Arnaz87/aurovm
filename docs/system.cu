#include "hip/hip_runtime.h"

/**< Add hoc utility module for working with the system */

/** File handle */
type file;

/** Prints a line to standard output */
void print (string);

/** Reads a line from standard input */
string read ();

/** Executes a command in the OS shell and returns its output */
string cmd (string);

/** Executes a command in the OS shell and returns its error code */
int exec (string);

/** Returns the time spent by the process in seconds */
float clock ();

/** Opens a file in the given mode */
file open (string path, string mode);

/** Reads all the contents of a file to string */
string readall (string path);

/** Writes a string to a file*/
void write (file, string);

/** Writes a single byte to a file */
void writebyte (file, int);
