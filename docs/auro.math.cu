#include "hip/hip_runtime.h"

float pi ();
float e ();
float sqrt2 ();

float abs (float);
float ceil (float);
float floor (float);
float round (float);
float trunc (float);

float ln (float);
float exp (float);
float sqrt (float);
float cbrt (float);
float pow (float base, float exponent);
float log (float, float base);
float mod (float numerator, float denominator);

float sin (float);
float cos (float);
float tan (float);
float asin (float);
float acos (float);
float atan (float);
float atan2 (float y, float x);
float sinh (float);
float cosh (float);
float tanh (float);
