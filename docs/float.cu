#include "hip/hip_runtime.h"

/** Floating pofloat number. */

type float;

// The numbers representable by this type must be at least of 4 digits of
// decimal precision from 0.001 to 1000, that is, it must be able to represent
// 0, 0.001, 0.002, 0.003 ... 10.01 ... 100.1 ... 998, 999, 1000
// This suggest floating point with 10 mantissa and 2 exponent bits,
// or a fixed point with 10 bits on both sides (plus sign for both)

// Arithmetic

/** Returns the negative complement of the number. */
float neg (float);

/** Performs float addition. */
float add (float, float);

/** Performs float subtraction. */
float sub (float, float);

/** Performs float multiplication. */
float mul (float, float);

/** Performs float division, rounding towards zero. */
float div (float, float);


// Comparisons

/** Tests wether an float is not zero. */
bool nz (float);

/** Tests wether an float is greater than zero. */
bool gz (float);

/** Tests wether two floats are equal. Algebraic identities might not hold after float calculations. */
bool eq (float, float);

/** Tests wether two floats are not equal. */
bool ne (float, float);

/** Tests wether the first float is greater than the second. */
bool gt (float, float);

/** Tests wether the first float is greater or equal than the second. */
bool ge (float, float);

/** Tests wether the first float is less than the second. */
bool lt (float, float);

/** Tests wether the first float is less or equal than the second. */
bool le (float, float);


// Creation

/** Converts an int to a float. */
float itof (int);

/** Converts a float to an int. */
int ftoi (float);

/** Creates a float given a magnitude and an exponent base 10 */
float decimal (int magnitude, int exponent);

float nan ();
float inf ();
bool isnan (float);
bool isinf (float);
