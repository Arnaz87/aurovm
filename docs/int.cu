#include "hip/hip_runtime.h"

/** Integer number. Every operation errors on overflow. */

type int;

/* This type is not appropiate for machine interpretations of integers,
    only for natural, human level arithmetic with small numbers,
    and is the most natural for the implementation.
   For integer types appropriate for machine manipulations,
    look for cobre.primitive. */

// These bounds suggest a minimum bit size of 16 bits

/** Maximum possible value for an int. It's value is platform dependent, but it's guaranteed to be greater or equal than  32000. */
int max;

/** Minimum possible value for an int. It's value is platform dependent, but it's guaranteed to be less or equal than -32000. */
int min;


// Arithmetic

/** Returns the negative complement of the number. */
int neg (int);

/** Performs integer addition. */
int add (int, int);

/** Performs integer subtraction. */
int sub (int, int);

/** Performs integer multiplication. */
int mul (int, int);

/** Performs integer division, rounding towards zero. */
int div (int, int);


// Comparisons

/** Tests wether an integer is not zero. */
bool nz (int);

/** Tests wether an integer is greater than zero. */
bool gz (int);

/** Tests wether two integers are equal. */
bool eq (int, int);

/** Tests wether two integers are not equal. */
bool ne (int, int);

/** Tests wether the first integer is greater than the second. */
bool gt (int, int);

/** Tests wether the first integer is greater or equal than the second. */
bool ge (int, int);

/** Tests wether the first integer is less than the second. */
bool lt (int, int);

/** Tests wether the first integer is less or equal than the second. */
bool le (int, int);

